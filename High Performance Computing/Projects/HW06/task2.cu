#include "scan.cuh"
#include <vector>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " n threads_per_block" << std::endl;
        return 1;
    }

    int n = std::atoi(argv[1]);
    int threads_per_block = std::atoi(argv[2]);

    // Allocate managed memory for input and output
    float *input, *output;
    hipMallocManaged(&input, n * sizeof(float));
    hipMallocManaged(&output, n * sizeof(float));

    // Fill the input array with random numbers
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);
    for (int i = 0; i < n; i++) {
        input[i] = dis(gen);
    }

    // Record the start event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, nullptr);
    // Execute the scan function
    scan(input, output, n, threads_per_block);
    hipDeviceSynchronize(); // Wait for compute device to finish
    hipEventRecord(stop, nullptr);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the last element of output array, which is the total sum for an inclusive scan
    std::cout << output[n-1] << std::endl;
    std::cout << milliseconds << std::endl;

    // Free memory and destroy events
    hipFree(input);
    hipFree(output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
