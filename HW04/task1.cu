#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "matmul.cuh"

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: ./task1 n threads_per_block" << std::endl;
        return 1;
    }

    int n = std::atoi(argv[1]);
    int threads_per_block = std::atoi(argv[2]);

    float* A = new float[n * n];
    float* B = new float[n * n];
    float* C = new float[n * n];

    for (int i = 0; i < n * n; i++) {
        A[i] = (rand() / (float)RAND_MAX) * 2.0f - 1.0f;
        B[i] = (rand() / (float)RAND_MAX) * 2.0f - 1.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_B, n * n * sizeof(float));
    hipMalloc(&d_C, n * n * sizeof(float));

    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul(d_A, d_B, d_C, n, threads_per_block);
    hipEventRecord(stop);

    hipMemcpy(C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << C[n * n - 1] << std::endl;
    std::cout << milliseconds << std::endl;

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
